
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecadd(float *out, float* a, float* b, int n) {
	for(int i = 0; i < n; i++) {
		out[i] = a[i] + b[i];
	}
}

int main() {
	const int N = 1000;
	float *a, *b, *out;
	float *d_a;
	a = (float*) malloc(1000*sizeof(float));
	hipMalloc((void**)&d_a, 1000*sizeof(float));
	hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
	vecadd<<<1,1>>>(out, d_a, b, 1000);
	hipFree(d_a);
	free(a);
}